
#include <hip/hip_runtime.h>
#include<iostream>

#define SIZE 1000

__global__ void vecSum(int* a,int*b,int* c){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i<SIZE){
		c[i] = a[i]+b[i];
	}
}

int main(){
	int h_a[SIZE],h_b[SIZE],h_c[SIZE];
	int* d_a,*d_b,*d_c;
	for(int i =0;i<SIZE;i++){
		h_a[i] = i*i;
		h_b[i] = i;
	}
	hipMalloc((void**)&d_a,SIZE*sizeof(int));
	hipMalloc((void**) &d_b,SIZE*sizeof(int));
	hipMalloc((void**) &d_c,SIZE*sizeof(int));
	hipMemcpy(d_a,h_a,SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,SIZE*sizeof(int), hipMemcpyHostToDevice);
	vecSum<<<ceil((float)SIZE/256),256>>>(d_a,d_b,d_c);
	hipMemcpy(h_c,d_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0;i<SIZE;i++){
		std::cout<<h_a[i]<<" + "<<h_b[i]<<" = "<<h_c[i]<<"\n";
	}
	std::cout<<'\n';
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}