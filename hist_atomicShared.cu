#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

#define NUM_BINS 7

__global__ void hist_kernel(char *data, int length, unsigned int *hist) {
    __shared__ unsigned int hist_s[NUM_BINS];

    // Initialize shared memory histogram bins
    if (threadIdx.x < NUM_BINS) {
        hist_s[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate the index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int ch = data[i] - 'a';
        if (ch >= 0 && ch < 26) {
            atomicAdd(&(hist_s[ch / 4]), 1);
        }
    }
    __syncthreads();

    // Copy results from shared memory to global memory
    if (threadIdx.x < NUM_BINS && hist_s[threadIdx.x] > 0) {
        atomicAdd(&(hist[threadIdx.x]), hist_s[threadIdx.x]);
    }
}

int main() {
    std::string data = "programming massively parallel processors";
    int length = data.length();

    // Allocate memory for histogram on the host
    unsigned int *hist;
    hist = (unsigned int*)malloc(NUM_BINS * sizeof(int));
    memset(hist, 0, NUM_BINS * sizeof(int));

    // Allocate memory on the device
    char *data_d;
    unsigned int *hist_d;
    hipMalloc((void**)&data_d, length * sizeof(char));
    hipMalloc((void**)&hist_d, NUM_BINS * sizeof(int));

    // Copy data to device
    hipMemcpy(data_d, data.c_str(), length * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(hist_d, 0, NUM_BINS * sizeof(int));

    // Define kernel launch parameters
    dim3 dimBlock(8, 1, 1);
    dim3 dimGrid((length + dimBlock.x - 1) / dimBlock.x, 1, 1);

    // Launch kernel
    hist_kernel<<<dimGrid, dimBlock>>>(data_d, length, hist_d);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(hist, hist_d, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

    // Print the histogram
    std::cout << "Histogram:\n";
    for (int i = 0; i < NUM_BINS; i++) {
        std::cout << hist[i] << " ";
    }

    // Free memory
    free(hist);
    hipFree(data_d);
    hipFree(hist_d);
    
    return 0;
}
