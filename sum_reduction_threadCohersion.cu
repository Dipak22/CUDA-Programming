#include <iostream>
#include <hip/hip_runtime.h>
#define LENGTH 2048
#define BLOCKDIM 64
#define COARSE_FACTOR 1

__global__ void sum_kernel(int *input, int *output) {
    int t = threadIdx.x;
    __shared__ int input_s[BLOCKDIM];
    int segment = 2*blockDim.x*blockIdx.x*COARSE_FACTOR;
    int index = segment+t;
    // Load data into shared memory with boundary check
    int sum = input[index];
    if(t<BLOCKDIM){
        for(unsigned int tile =1;tile<2*COARSE_FACTOR;++tile){
            sum += input[index + tile*BLOCKDIM];
        input_s[t] = sum;

        }
    }

    // Perform reduction within shared memory
    for (unsigned int stride = BLOCKDIM / 2; stride > 0; stride >>=1) {
        __syncthreads(); 
        if (t < stride) {
           // printf("%d ",index);
            input_s[t] += input_s[t + stride];
        }
        //printf("\n");
         // Ensure all threads have completed current stride
    }

    // Write the result to output
    if (t == 0) {
        atomicAdd(output,input_s[0]);
    }
}

int main() {
    int *input, *sum;
    input = (int *)malloc(LENGTH * sizeof(int));
    sum = (int *)malloc(sizeof(int));

    // Initialize input data
    for (int i = 0; i < LENGTH; i++) {
        input[i] = rand() % 10;
    }

    // Allocate memory on the device
    int *input_d, *output;
    hipMalloc((void **)&input_d, LENGTH * sizeof(int));
    hipMalloc((void **)&output, sizeof(int));

    // Copy input data to the device
    hipMemcpy(input_d, input, LENGTH * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with BLOCKDIM threads
    dim3 dimBlock(BLOCKDIM, 1, 1);
    dim3 dimGrid((LENGTH+BLOCKDIM-1)/BLOCKDIM, 1, 1);
    sum_kernel<<<dimGrid, dimBlock>>>(input_d, output);

    // Copy the result back to the host
    hipMemcpy(sum, output, sizeof(int), hipMemcpyDeviceToHost);

    // Display the input array and the GPU-computed sum
    std::cout << "Input:\n";
    for (int i = 0; i < LENGTH; i++) {
        std::cout << input[i] << " ";
    }
    std::cout << "\nGPU Sum: " << *sum;

    // Calculate and display the CPU sum for verification
    int cpu_sum = 0;
    for (int i = 0; i < LENGTH; i++) {
        cpu_sum += input[i];
    }
    std::cout << "\nCPU Sum: " << cpu_sum << "\n";

    // Free memory
    free(input);
    free(sum);
    hipFree(input_d);
    hipFree(output);

    return 0;
}
