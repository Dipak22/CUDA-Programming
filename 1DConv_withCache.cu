
#include <hip/hip_runtime.h>
#include<iostream>
#define SIZE 50
#define FILTER_RADIUS 2
__constant__ float F_d[2*FILTER_RADIUS+1];

__global__ void conv1d_Basic(float* N,float* P){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    float pValue = 0;
    if(i<SIZE){
        for(int j=-FILTER_RADIUS;j<=FILTER_RADIUS;j++){
            if(i+j>=0 && i+j<SIZE){
                pValue += N[i+j]*F_d[FILTER_RADIUS+j];
            }
        }
        P[i] = pValue;
    }
    
}

int main(){
    float* N,*P,*F;
    N = (float*)malloc(SIZE*sizeof(float));
    P = (float*)malloc(SIZE* sizeof(float));
    F = (float*)malloc((2*FILTER_RADIUS+1)*sizeof(float));
    float *N_d,*P_d;
    for(int i=0;i<SIZE;i++){
        N[i] = rand()%10;
    }
    for(int i=0;i<2*FILTER_RADIUS+1;i++)
        F[i] = rand()%10;
    hipMalloc((void**)&N_d,SIZE*sizeof(float));
    hipMalloc((void**)&P_d,SIZE*sizeof(float));
    hipMemcpy(N_d,N,SIZE*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(F_d),F,(2*FILTER_RADIUS+1)*sizeof(float));
    dim3 dimGrid(ceil((float)SIZE/32.0),1,1);
    dim3 dimBlock(32,1,1);
    conv1d_Basic<<<dimGrid,dimBlock>>>(N_d,P_d);
    hipMemcpy(P,P_d,SIZE*sizeof(float),hipMemcpyDeviceToHost);
    std::cout<<"Input : \n";
    for(int i=0;i<SIZE;i++)
        std::cout<<N[i]<<" ";
    std::cout<<"\nFilter: \n";
    for(int i=0;i<2*FILTER_RADIUS+1;i++)
        std::cout<<F[i]<<" ";
    std::cout<<"\nOutput : \n";
    for(int i=0;i<SIZE;i++)
        std::cout<<P[i]<<" ";
    free(N);
    free(P);
    free(F);
    hipFree(N_d);
    //cudaFree(F_d);
    hipFree(P_d);
    return 0;
}